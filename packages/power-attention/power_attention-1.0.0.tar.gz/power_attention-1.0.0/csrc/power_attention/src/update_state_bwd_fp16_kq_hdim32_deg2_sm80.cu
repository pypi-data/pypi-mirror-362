// Copyright (c) 2024. Sean Zhang.
// Splitting the different degrees to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "update_state_launch_template.h"

template<>
void run_compute_update_states_bwd<cutlass::half_t, 32, 2>(Update_state_bwd_params &params, hipStream_t stream) {
    run_compute_update_states_bwd_<cutlass::half_t, 32, 2>(params, stream);
}
