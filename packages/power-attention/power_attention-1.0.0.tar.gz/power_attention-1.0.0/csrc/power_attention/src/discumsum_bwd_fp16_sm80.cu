// Copyright (c) 2024. Sean Zhang.
// Splitting the different degrees to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "discumsum_launch_template.h"

template<>
void run_discumsum_bwd<cutlass::half_t>(Discumsum_bwd_params &params, hipStream_t stream) {
    run_discumsum_bwd_<cutlass::half_t>(params, stream);
}
