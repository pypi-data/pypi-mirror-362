// Copyright (c) 2024, Sean Zhang.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "power_fwd_launch_template.h"

template<>
void run_mha_fwd_<cutlass::half_t, 64, 2, true>(Power_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim64<cutlass::half_t, 2, true>(params, stream);
}
