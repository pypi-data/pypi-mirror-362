// Copyright (c) 2024, Sean Zhang.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "power_bwd_launch_template.h"

template<>
void run_mha_bwd_<cutlass::half_t, 32, 1>(Power_bwd_params &params, hipStream_t stream) {
    run_mha_bwd_hdim32<cutlass::half_t, 1>(params, stream);
}
