// Copyright (c) 2024, Sean Zhang.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "power_fwd_launch_template.h"

template<>
void run_mha_fwd_<cutlass::bfloat16_t, 32, 4, true>(Power_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim32<cutlass::bfloat16_t, 4, true>(params, stream);
}
