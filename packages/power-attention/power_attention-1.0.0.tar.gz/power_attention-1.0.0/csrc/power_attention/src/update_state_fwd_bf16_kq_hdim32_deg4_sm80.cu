// Copyright (c) 2024. Sean Zhang.
// Splitting the different degrees to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "update_state_launch_template.h"

template<>
void run_compute_update_states<cutlass::bfloat16_t, 32, 4>(Update_state_params &params, hipStream_t stream) {
    run_update_states_fwd_<cutlass::bfloat16_t, 32, 4>(params, stream);
}
