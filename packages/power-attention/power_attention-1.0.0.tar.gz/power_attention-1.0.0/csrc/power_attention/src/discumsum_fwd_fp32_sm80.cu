// Copyright (c) 2024. Sean Zhang.
// Splitting the different degrees to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "discumsum_launch_template.h"

template<>
void run_discumsum_fwd<float>(Discumsum_params &params, hipStream_t stream) {
    run_discumsum_fwd_<float>(params, stream);
}
