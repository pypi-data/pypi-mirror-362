// Copyright (c) 2024. Sean Zhang.
// Splitting the different degrees to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "discumsum_launch_template.h"

template<>
void run_discumsum_fwd<cutlass::bfloat16_t>(Discumsum_params &params, hipStream_t stream) {
    run_discumsum_fwd_<cutlass::bfloat16_t>(params, stream);
}
