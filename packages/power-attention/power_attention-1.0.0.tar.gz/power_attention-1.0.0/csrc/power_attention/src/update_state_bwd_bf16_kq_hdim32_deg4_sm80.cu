// Copyright (c) 2024. Sean Zhang.
// Splitting the different degrees to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "update_state_launch_template.h"

template<>
void run_compute_update_states_bwd<cutlass::bfloat16_t, 32, 4>(Update_state_bwd_params &params, hipStream_t stream) {
    run_compute_update_states_bwd_<cutlass::bfloat16_t, 32, 4>(params, stream);
}
